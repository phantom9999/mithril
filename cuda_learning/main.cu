#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

int main() {
    int count = 0;
    hipGetDeviceCount(&count);
    std::cout << "gpu count: " << count << std::endl;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "device number: " << 0 << std::endl;
    std::cout << "name: " << prop.name << std::endl;
    std::cout << "memory: " << prop.totalGlobalMem << std::endl;
    std::cout << "threads per block: " << prop.maxThreadsPerBlock << std::endl;
  return 0;
}
