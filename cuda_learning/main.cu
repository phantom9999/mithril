#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

int main() {
    int count = 0;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
        return 0;
    }
    std::cout << "gpu count: " << count << std::endl;
    for (int i=0; i<count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "device number: " << 0 << std::endl;
        std::cout << "name: " << prop.name << std::endl;
        std::cout << "memory: " << prop.totalGlobalMem << std::endl;
        std::cout << "threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "compute: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "ms count: " << prop.multiProcessorCount << std::endl;
        std::cout << "thread wrap size: " << prop.warpSize << std::endl;
    }

  return 0;
}
